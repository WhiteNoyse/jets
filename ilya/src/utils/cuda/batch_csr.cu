#include "hip/hip_runtime.h"
#define BATCH_SIZE ${BATCH_SIZE}
#define NUM_ROWS ${NUM_ROWS}

extern "C"
__global__ void batch_csr(const long *indices, long *col_ind, long *col_ptr, long nnz) {
  long ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (!(ind < nnz)) {
    return;
  }
  long batch_id = indices[ind];
  long row_id = indices[ind + nnz];
  long col_id = indices[ind + 2 * nnz];

  long prev_batch_id = -1;
  if (ind > 0) {
    prev_batch_id = indices[ind - 1];
  }

  long prev_row_id = -1;
  if (ind > 0) {
    prev_row_id = indices[ind - 1 + nnz];
  }

  if (ind < nnz) {
    col_ind[ind] = col_id;
  }

  if ((batch_id != prev_batch_id) || (row_id != prev_row_id)) {
    col_ptr[batch_id * (NUM_ROWS + 1) + row_id] = ind;

    if (batch_id > 0 && row_id == 0) {
      col_ptr[prev_batch_id * (NUM_ROWS + 1) + prev_row_id + 1] = ind;
    }
  }

  if (ind + 1 == nnz) {
    col_ptr[batch_id * (NUM_ROWS + 1) + row_id + 1] = ind + 1;
  }
}
